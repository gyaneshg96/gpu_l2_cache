
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

void vec_add(double* c, const double* a, const double* b, long N){
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    c[i] = a[i] + b[i];
  }
}

__global__
void vec_add_kernel(double* c, const double* a, const double* b, long N){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) c[idx] = a[idx] + b[idx];
}

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

int main() {
  long N = (1UL<<25);

  double *x, *y, *z;
  hipHostMalloc((void**)&x, N * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&y, N * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&z, N * sizeof(double), hipHostMallocDefault);
  double* z_ref = (double*) malloc(N * sizeof(double));
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    x[i] = i+2;
    y[i] = 1.0/(i+1);
    z[i] = 0;
    z_ref[i] = 0;
  }

  double tt = omp_get_wtime();
  vec_add(z_ref, x, y, N);
  printf("CPU Bandwidth = %f GB/s\n", 3*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double *x_d, *y_d, *z_d;
  hipMalloc(&x_d, N*sizeof(double));
  hipMalloc(&y_d, N*sizeof(double));
  hipMalloc(&z_d, N*sizeof(double));

  tt = omp_get_wtime();
  hipMemcpyAsync(x_d, x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(y_d, y, N*sizeof(double), hipMemcpyHostToDevice);
  vec_add_kernel<<<N/1024+1,1024>>>(z_d, x_d, y_d, N);
  hipMemcpyAsync(z, z_d, N*sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("GPU Bandwidth = %f GB/s\n", 3*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double err = 0;
  for (long i = 0; i < N; i++) err += fabs(z[i]-z_ref[i]);
  printf("Error = %f\n", err);

  hipFree(x_d);
  hipFree(y_d);
  hipFree(z_d);

  hipHostFree(x);
  hipHostFree(y);
  hipHostFree(z);
  free(z_ref);

  return 0;
}

